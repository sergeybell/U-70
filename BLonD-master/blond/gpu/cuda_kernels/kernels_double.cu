#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>
#include <hiprand/hiprand_kernel.h>
#define PI 3.141592653589793238462643383279502884197169399375105820974944592307816406286
#define PI_DIV_2 3.141592653589793238462643383279502884197169399375105820974944592307816406286/2

// Note that any atomic operation can be implemented based on atomicCAS() (Compare And Swap).
// For example, atomicAdd() for double-precision floating-point numbers is not
// available on devices with compute capability lower than 6.0 but it can be implemented
// as follows:
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

extern "C"
__global__ void simple_kick(
    double  * __restrict__ beam_dt,
    double        * __restrict__ beam_dE,
    const int n_rf,
    const double  * __restrict__ voltage,
    const double  * __restrict__ omega_RF,
    const double  * __restrict__ phi_RF,
    const int n_macroparticles,
    const double acc_kick
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double my_beam_dt;
    double my_beam_dE;
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        my_beam_dt = beam_dt[i];
        my_beam_dE = beam_dE[i];
        for (int j = 0; j < n_rf; j++) {
            my_beam_dE += voltage[j] * sin(omega_RF[j]*my_beam_dt + phi_RF[j]);
        }
        beam_dE[i] = my_beam_dE + acc_kick;
    }
}

extern "C"
__global__ void rf_volt_comp(const double * __restrict__ voltage,
                             const double * __restrict__ omega_rf,
                             const double * __restrict__ phi_rf,
                             const double * __restrict__ bin_centers,
                             const int n_rf,
                             const int n_bins,
                             double * __restrict__ rf_voltage)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double my_rf_voltage;
    double my_bin_centers;
    for (int i = tid; i < n_bins; i += blockDim.x * gridDim.x) {
        my_rf_voltage = rf_voltage[i];
        my_bin_centers = bin_centers[i];
        for (int j = 0; j < n_rf; j++)
            my_rf_voltage += voltage[j] * sin(omega_rf[j] * my_bin_centers + phi_rf[j]);
        rf_voltage[i] = my_rf_voltage;
    }
}

extern "C"
__global__ void drift(double * __restrict__ beam_dt,
                     double  * __restrict__ beam_dE,
                     const int solver,
                     const double T0, const double length_ratio,
                     const double alpha_order, const double eta_zero,
                     const double eta_one, const double eta_two,
                     const double alpha_zero, const double alpha_one,
                     const double alpha_two,
                     const double beta, const double energy,
                     const int n_macroparticles)
{
    double T = T0 * length_ratio;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if ( solver == 0 )
    {
        double coeff = eta_zero / (beta * beta * energy);
        for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
            beam_dt[i] += T * coeff * beam_dE[i];
    }

    else if ( solver == 1 )
    {
        const double coeff = 1. / (beta * beta * energy);
        const double eta0 = eta_zero * coeff;
        const double eta1 = eta_one * coeff * coeff;
        const double eta2 = eta_two * coeff * coeff * coeff;

        if (alpha_order == 0)
            for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
                beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]) - 1.);
        else if (alpha_order == 1)
            for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
                beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]
                                         - eta1 * beam_dE[i] * beam_dE[i]) - 1.);
        else
            for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
                beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]
                                         - eta1 * beam_dE[i] * beam_dE[i]
                                         - eta2 * beam_dE[i] * beam_dE[i] * beam_dE[i]) - 1.);
    }

    else
    {

        const double invbetasq = 1 / (beta * beta);
        const double invenesq = 1 / (energy * energy);
        // double beam_delta;


        for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)

        {

            double beam_delta = sqrt(1. + invbetasq *
                              (beam_dE[i] * beam_dE[i] * invenesq + 2.*beam_dE[i] / energy)) - 1.;

            beam_dt[i] += T * (
                              (1. + alpha_zero * beam_delta +
                               alpha_one * (beam_delta * beam_delta) +
                               alpha_two * (beam_delta * beam_delta * beam_delta)) *
                              (1. + beam_dE[i] / energy) / (1. + beam_delta) - 1.);
        }
    }
}


extern "C"
__global__ void hybrid_histogram(const double * __restrict__  input,
                                 double * __restrict__  output, const double cut_left,
                                 const double cut_right, const unsigned int n_slices,
                                 const int n_macroparticles, const int capacity)
{
    extern __shared__ int block_hist[];
    //reset shared memory
    for (int i = threadIdx.x; i < capacity; i += blockDim.x)
        block_hist[i] = 0;
    __syncthreads();
    int const tid = threadIdx.x + blockDim.x * blockIdx.x;
    int target_bin;
    double const inv_bin_width = n_slices / (cut_right - cut_left);

    const int low_tbin = (n_slices / 2) - (capacity / 2);
    const int high_tbin = low_tbin + capacity;


    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        target_bin = floor((input[i] - cut_left) * inv_bin_width);
        if (target_bin < 0 || target_bin >= n_slices)
            continue;
        if (target_bin >= low_tbin && target_bin < high_tbin)
            atomicAdd(&(block_hist[target_bin - low_tbin]), 1);
        else
            atomicAdd(&(output[target_bin]), 1);

    }
    __syncthreads();
    for (int i = threadIdx.x; i < capacity; i += blockDim.x)
        atomicAdd(&output[low_tbin + i], (double) block_hist[i]);
}


extern "C"
__global__ void sm_histogram(const double * __restrict__  input,
                             double * __restrict__  output, const double cut_left,
                             const double cut_right, const unsigned int n_slices,
                             const int n_macroparticles)
{
    extern __shared__ int block_hist[];
    for (int i = threadIdx.x; i < n_slices; i += blockDim.x)
        block_hist[i] = 0;
    __syncthreads();
    int const tid = threadIdx.x + blockDim.x * blockIdx.x;
    int target_bin;
    double const inv_bin_width = n_slices / (cut_right - cut_left);
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        target_bin = floor((input[i] - cut_left) * inv_bin_width);
        if (target_bin < 0 || target_bin >= n_slices)
            continue;
        atomicAdd(&(block_hist[target_bin]), 1);
    }
    __syncthreads();
    for (int i = threadIdx.x; i < n_slices; i += blockDim.x)
        atomicAdd(&output[i], (double) block_hist[i]);
}


extern "C"
__global__ void lik_only_gm_copy(
    double * __restrict__ beam_dt,
    double * __restrict__ beam_dE,
    const double * __restrict__ voltage_array,
    const double * __restrict__ bin_centers,
    const double charge,
    const int n_slices,
    const int n_macroparticles,
    const double acc_kick,
    double * __restrict__ glob_vkick_factor
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);


    for (int i = tid; i < n_slices - 1; i += gridDim.x * blockDim.x) {
        glob_vkick_factor[2*i] = charge * (voltage_array[i + 1] - voltage_array[i])
                              * inv_bin_width;
        glob_vkick_factor[2*i+1] = (charge * voltage_array[i] - bin_centers[i] * glob_vkick_factor[2*i])
                         + acc_kick;
    }
}


extern "C"
__global__ void lik_only_gm_comp(
    double * __restrict__ beam_dt,
    double * __restrict__ beam_dE,
    const double * __restrict__ voltage_array,
    const double * __restrict__ bin_centers,
    const double charge,
    const int n_slices,
    const int n_macroparticles,
    const double acc_kick,
    double * __restrict__ glob_vkick_factor
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);
    int fbin;
    const double bin0 = bin_centers[0];
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        fbin = floor((beam_dt[i] - bin0) * inv_bin_width);
        if ((fbin < n_slices - 1) && (fbin >= 0))
            beam_dE[i] += beam_dt[i] * glob_vkick_factor[2*fbin] + glob_vkick_factor[2*fbin+1];
    }
}


extern "C"
__global__ void lik_drift_only_gm_comp(
    double *beam_dt,
    double *beam_dE,
    const double *voltage_array,
    const double *bin_centers,
    const double charge,
    const int n_slices,
    const int n_macroparticles,
    const double acc_kick,
    double *glob_vkick_factor,
    const double T0, const double length_ratio,
    const double eta0, const double beta, const double energy
)
{
    const double T = T0 * length_ratio * eta0 / (beta * beta * energy);

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);
    unsigned fbin;
    const double bin0 = bin_centers[0];
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        fbin = (unsigned) floor((beam_dt[i] - bin0) * inv_bin_width);
        if ((fbin < n_slices - 1))
            beam_dE[i] += beam_dt[i] * glob_vkick_factor[2*fbin] + glob_vkick_factor[2*fbin+1];
        // beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]) -1.);
        beam_dt[i] += T * beam_dE[i];
    }
}

// This function calculates and applies only the synchrotron radiation damping term
extern "C"
__global__ void synchrotron_radiation(
    double *  beam_dE,
    const double U0,
    const int n_macroparticles,
    const double tau_z,
    const int n_kicks)
{

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    
    // SR damping constant, adjusted for better performance
    const double const_synch_rad = 1.0 - 2.0 / tau_z;

    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        // SR damping term due to energy spread and
        // Average energy change due to SR
        for (int j = 0; j < n_kicks; j++) {
            beam_dE[i] = beam_dE[i] * const_synch_rad - U0;
        }
    }
}


// This function calculates and applies synchrotron radiation damping and
// quantum excitation terms
extern "C"
__global__ void synchrotron_radiation_full(
    double *  beam_dE,
    const double U0,
    const int n_macroparticles,
    const double sigma_dE,
    const double tau_z,
    const double energy,
    const int n_kicks
)
{
    unsigned int seed = 1234;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    // Quantum excitation constant
    const double const_quantum_exc = 2.0 * sigma_dE / sqrt(tau_z) * energy;
    
    // Adjusted SR damping constant
    const double const_synch_rad = 1.0 - 2.0 / tau_z;

    hiprandState_t state;
    hiprand_init(seed, tid, 0, &state);

    // Compute synchrotron radiation damping term and
    // Applies the quantum excitation term
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_kicks; j++) {
            beam_dE[i] = beam_dE[i] * const_synch_rad 
                         + const_quantum_exc * hiprand_normal_double(&state) - U0;
        }
    }
}


// extern "C"
// __global__ void cuinterp(double *x,
//                          int x_size,
//                          double *xp,
//                          int xp_size,
//                          double *yp,
//                          double *y,
//                          double left,
//                          double right)
// {
//     if (left == 0.12345)
//         left = yp[0];
//     if (right == 0.12345)
//         right = yp[xp_size - 1];
//     double curr;
//     int lo;
//     int mid;
//     int hi;
//     int tid = threadIdx.x + blockDim.x * blockIdx.x;
//     for (int i = tid; i < x_size; i += blockDim.x * gridDim.x) {
//         //need to find the right bin with binary search
//         // looks like bisect_left
//         curr = x[i];
//         hi = xp_size;
//         lo = 0;
//         while (lo < hi) {
//             mid = (lo + hi) / 2;
//             if (xp[mid] < curr)
//                 lo = mid + 1;
//             else
//                 hi = mid;
//         }
//         if (lo == xp_size)
//             y[i] = right;
//         else if (xp[lo - 1] == curr)
//             y[i] = yp[i];
//         else if (lo <= 1)
//             y[i] = left;
//         else {
//             y[i] = yp[lo - 1] +
//                    (yp[lo] - yp[lo - 1]) * (x[i] - xp[lo - 1]) /
//                    (xp[lo] - xp[lo - 1]);
//         }

//     }
// }

// extern "C"
// __global__ void cugradient(
//     double x,
//     int *y,
//     double *g,
//     int size)
// {
//     int tid = threadIdx.x + blockDim.x * blockIdx.x;
//     for (int i = tid + 1; i < size - 1; i += blockDim.x * gridDim.x) {

//         g[i] = (y[i + 1] - y[i - 1]) / (2 * x);
//         // g[i] = (hs*hs*fd + (hd*hd-hs*hs)*fx - hd*hd*fs)/
//         //     (hs*hd*(hd+hs));
//     }
//     if (tid == 0)
//         g[0] = (y[1] - y[0]) / x;
//     if (tid == 32)
//         g[size - 1] = (y[size - 1] - y[size - 2]) / x;
// }
